#include "hip/hip_runtime.h"
﻿#include ""

#include "CpuGpuMem.h"
#include "KernelGpuAdd.cuh"

__global__ void gpu_add(int* gpu_numbers, const int nc)
{
	int id = blockDim.x * blockIdx.x + threadIdx.x;

	if (id < nc)
		gpu_numbers[id] *= 2;
}

void cpuGpuExecute(CpuGpuMem* cg)
{
	int number_count = cg->allocSize;

	int blockDim = 64;
	int gridDim = (number_count + blockDim - 1) / blockDim;

	//execute
	for (size_t i = 0; i < 4; i++)
		gpu_add << <gridDim, blockDim, 0, cg->stream >> > ((int*)cg->gpuPtr, number_count);
}

#include "hip/hip_runtime.h"
#include <windows.h>
#include <cmath>

#include ""

#include "CpuGpuMem.h"
#include "KernelGpuAdd.cuh"
#include "hip/hip_runtime.h"

#define BIAS 1

__global__ void conv1GPU(float* resultImages, float* masks, int* image, int width, int height, int maskSize, int rMatrixWidth, int rMatrixHeight, int maskCount)
{
	int id = blockDim.x * blockIdx.x + threadIdx.x;

	if(id < maskCount * rMatrixWidth * rMatrixHeight ) {
		int temp = 0;
		int j = id % rMatrixWidth;
		temp = id / rMatrixWidth;
		int i = temp % rMatrixHeight;
		int m = temp / rMatrixHeight;

		for (int k = 0; k < maskSize * maskSize; k++) {
			int mCol = k % maskSize;
			int mRow = k / maskSize;
			resultImages[(m * rMatrixWidth * rMatrixHeight) + i * rMatrixWidth + j] +=
				(float)image[(width * i + j) + mRow * width + mCol] * masks[m * (maskSize * maskSize) + k];
		}
		resultImages[(m * rMatrixWidth * rMatrixHeight) + i * rMatrixWidth + j] += (float)BIAS * masks[maskCount * (maskSize * maskSize) + m];
	}
}

__global__ void batchConv1GPU(float* feature, float* batchWeights, int width, int height, int featureCount)
{
	int id = blockDim.x * blockIdx.x + threadIdx.x;

	if (id < featureCount * width * height) {
		int i = id % (width * height);
		int m = id / (width * height);

		float sDeviation = 0.0; // standart sapma i�in

		sDeviation = sqrt(batchWeights[(featureCount * 3) + m]);

		feature[(m * width * height) + i] = (feature[(m * width * height) + i] - batchWeights[featureCount * 2 + m]) / sDeviation;
		feature[(m * width * height) + i] = feature[(m * width * height) + i] * batchWeights[m] + batchWeights[featureCount + m];

		if (fabs(feature[(m * width * height) + i]) + feature[(m * width * height) + i] < 0.01) {
			feature[(m * width * height) + i] = 0.0;
		}
	}
}

void batchAndReLuConv1ExecGPU(CpuGpuMem* cg, const int featureCount)
{
	int ms = cg->maskWHSize;
	int fws = cg->featureWidthSize;
	int fhs = cg->featureHeightSize;

	int blockDim = 1024;
	int threadCount = featureCount * fws * fhs;


	int gridDim = (threadCount + blockDim - 1) / blockDim;

	batchConv1GPU << <gridDim, blockDim, 0, cg->stream >> > (cg->gpuFeaturePtr, cg->gpuBatchPtr, cg->featureWidthSize, cg->featureHeightSize, featureCount);

}

void conv1ExecGPU(CpuGpuMem* cg, const int maskCount)
{
	int iws = cg->imageWidthSize;
	int ihs = cg->imageHeightSize;
	int ms = cg->maskWHSize;
	int fws = cg->featureWidthSize = iws - ms + 1;
	int fhs = cg->featureHeightSize = ihs - ms + 1;

	int blockDim = 1024;
	int threadCount = maskCount * fws * fhs;


	int gridDim = (threadCount + blockDim - 1) / blockDim;

	
	conv1GPU << <gridDim, blockDim, 0 ,cg->stream>> > (cg->gpuFeaturePtr, cg->gpuMaskPtr, (int*)cg->gpuImagePtr, cg->imageWidthSize, cg->imageHeightSize,
		cg->maskWHSize,cg->featureWidthSize, cg->featureHeightSize, maskCount);

}




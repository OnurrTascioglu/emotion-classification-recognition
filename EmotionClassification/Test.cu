#include "hip/hip_runtime.h"
#include <windows.h>
#include <cmath>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>
#include ""
#include <time.h>
#include "CpuGpu.h"
#include "CpuGpuMem.h"
#include "KernelGpuAdd.cuh"
#include "hip/hip_runtime.h"

#define BIAS 1

//----------------conv1
__global__ void conv1GPU(int* image, float* resultImages, float* masks, int width, int height, int maskSize, int rMatrixWidth, int rMatrixHeight, int maskCount)
{
	int id = blockDim.x * blockIdx.x + threadIdx.x; //bloklardaki threadlar belirlenip idleri tan�mlan�r

	//Grafik kart�nda olu�turulan threadler warplara (32�lik thread b�l�mleri) ba��ml�
	//olu�turuldu�u i�in 32�nin kat� olmayan durumlarda fazladan thread olu�turulur.

	if (id < maskCount * rMatrixWidth * rMatrixHeight) { //��k�� n�ronlar� hesapland�ktan sonra Fazlal�k threadlerin ko�mamas� i�in
		int temp = 0;
		int j = id % rMatrixWidth;		//conv1() fonksiyonunda evri�im i�lemini yapan ana for d�ng�s�ndeki  j,i,m de�erlerinin bulunmas� i�in
		temp = id / rMatrixWidth;		//buradaki mod i�lemleri yap�l�r. i, j, m de�erleri threadin kendi id'sindeki for iterasyonunu bulmas�n� sa�lar.
		int i = temp % rMatrixHeight;
		int m = temp / rMatrixHeight;
		float tempSum = 0.0;

		for (int k = 0; k < maskSize * maskSize; k++) { // evri�im i�lemi burada yap�l�r conv1() fonksiyonu ile ayn� i�lemi yapar
			int mCol = k % maskSize;    //maske i�inde gezebilmek i�in mCol ve mRow de�erleri hesaplan�r.
			int mRow = k / maskSize;
			tempSum +=
				(float)image[(width * i + j) + mRow * width + mCol] * masks[m * (maskSize * maskSize) + k];  //maske gezdirme i�lemi
		}
		resultImages[(m * rMatrixWidth * rMatrixHeight) + i * rMatrixWidth + j] =  tempSum + (float)BIAS * masks[maskCount * (maskSize * maskSize) + m]; //Maske i�leminden sonra bias de�eri ile �arp�l�p toplan�r.
	}
}


__global__ void convHiddenGPU(float* feature, float* resultImages, float* weights, int fWidth, int fHeight, int maskSize, int maskCount, int maskDim)
{
	int id = blockDim.x * blockIdx.x + threadIdx.x; //bloklardaki threadlar belirlenip idleri tan�mlan�r

	int rMatrixWidth = fWidth - maskSize + 1;  //padding olmadan ��k�� katman�n�n y�kseklik ve geni�li�i hesaplan�r
	int rMatrixHeight = fHeight - maskSize + 1;

	//Grafik kart�nda olu�turulan threadler warplara (32�lik thread b�l�mleri) ba��ml�
	//olu�turuldu�u i�in 32�nin kat� olmayan durumlarda fazladan thread olu�turulur.

	if (id < maskCount * rMatrixWidth * rMatrixHeight) {  //��k�� n�ronlar� hesapland�ktan sonra Fazlal�k threadlerin ko�mamas� i�in
		int temp = 0;
		int j = id % rMatrixWidth;		//convHidden() fonksiyonunda evri�im i�lemini yapan ana for d�ng�s�ndeki  j,i,c de�erlerinin bulunmas� i�in	
		temp = id / rMatrixWidth;		//buradaki mod i�lemleri yap�l�r. i, j, c de�erleri threadin kendi id'sindeki for iterasyonunu bulmas�n� sa�lar.
		int i = temp % rMatrixHeight;
		int c = temp / rMatrixHeight;
		float tempSum = 0.0;

		// evri�im i�lemi burada yap�l�r conv1() fonksiyonu ile ayn� i�lemi yapar
		for (int d = 0; d < maskDim; d++) {  // maskenin derinlinin hesaplanmas� i�in
			for (int k = 0; k < maskSize * maskSize; k++) {  
				int mCol = k % maskSize;    // maske i�inde gezebilmek i�in mCol ve mRow de�erleri hesaplan�r.
				int mRow = k / maskSize;
				tempSum += (float)feature[d * fWidth * fHeight + (fWidth * i + j) + mRow * fWidth + mCol] * weights[c * (maskDim * maskSize * maskSize) + d * maskSize * maskSize + k]; //maske gezdirme i�lemi
			}
		}
		resultImages[(c * rMatrixWidth * rMatrixHeight) + i * rMatrixWidth + j] = tempSum + BIAS * weights[maskCount * maskDim * maskSize * maskSize + c]; //Maske i�leminden sonra bias de�eri ile �arp�l�p toplan�r.
	}
}


__global__ void batchNormGPU(float* feature, float* batchWeights, int width, int height, int featureCount)
{
	int id = blockDim.x * blockIdx.x + threadIdx.x; //bloklardaki threadlar belirlenip idleri tan�mlan�r

	if (id < featureCount * width * height) {
		int i = id % (width * height); //threadin kendi id'sindeki for iterasyonunu bulmas�n� sa�lar.
		int m = id / (width * height);

		float sDeviation = 0.0; // standart sapma i�in

		sDeviation = sqrt(batchWeights[(featureCount * 3) + m]); //varyans kullan�larak standart sapma hesaplan�r. (featureCount * 3) dizide varyans elemanlar�na eri�ir

		feature[(m * width * height) + i] = (feature[(m * width * height) + i] - batchWeights[featureCount * 2 + m]) / sDeviation; //Her bir de�er aritmetik ortalamadan ��kar�l�p standart sapmaya b�l�n�r. (featureCount * 2) aritmetik ortalama de�erlerine eri�ir
		feature[(m * width * height) + i] = feature[(m * width * height) + i] * batchWeights[m] + batchWeights[featureCount + m];  //Sonu� gamma ile �arp�l�r beta ile toplan�r.

		if (fabs(feature[(m * width * height) + i]) + feature[(m * width * height) + i] < 0.001) { 
			feature[(m * width * height) + i] = 0.0; //ReLU i�lemi
		}
	}
}

__global__ void maxPoolingGPU(float* feature, float* tempFeature, int width, int height, int  featureCount, int pool, int stride)
{
	int id = blockDim.x * blockIdx.x + threadIdx.x;  //bloklardaki threadlar belirlenip idleri tan�mlan�r


	if (id < featureCount * (width / stride) * (height / stride)) {
		int temp2 = 0;
		int col = id % (width / stride);   //threadin kendi id'sindeki for iterasyonunu bulmas�n� sa�lar.
		temp2 = id / (width / stride);
		int row = temp2 % (height / stride);
		int m = temp2 / (height / stride);

		float max = 0.0;
		float temp = 0.0;

		for (int k = 0; k < pool; k++) {  //pool*pool kadarl�k alandaki de�erlerden en y�ksek olan se�ilmelidir.
			for (int n = 0; n < pool; n++) {
				temp = feature[(m * width * height) + row * width * stride + col * stride + k * width + n]; //Pool �er�evesinin denk geldi�i feature de�erleri temp e atan�r.
				if ((temp - max) > 0.00001) {
					max = temp; //max de�er hesaplan�r.
				}
			}
		}
		tempFeature[(m * (width / stride) * (height / stride)) + (row * (width / stride)) + col] = max;  //bellek alan�ndan tasarruf amac�yla yeni dizi a�mak yerine, max de�erler feature dizisine atan�r.

	}

}

__global__ void flattenGPU(float* features, float* flattenArray, int width, int height, int featureCount) {
	int id = blockDim.x * blockIdx.x + threadIdx.x; //bloklardaki threadlar belirlenip idleri tan�mlan�r

	if (id < featureCount * width * height) {

		int temp = 0;
		int f = id % featureCount;  //threadin kendi id'sindeki for iterasyonunu bulmas�n� sa�lar.
		temp = id / featureCount;
		int j = temp % width;
		int i = temp / width;

		flattenArray[id] = features[f * width * height + i * width + j];  //flatten i�lemi burada yap�l�r. Giri� n�ronlar� a��rl�klara uygun gelecek �ekilde s�ralan�r.
	}
}

__global__ void denseGPU(float* inputLayer, float* outputLayer, float* weights, int inputLayerSize, int outputLayerSize) {
	int id = blockDim.x * blockIdx.x + threadIdx.x; //bloklardaki threadlar belirlenip idleri tan�mlan�r

	if (id < outputLayerSize) {
		// optimize edilmeli
		float tempSum = 0.0;

		for (int j = 0; j < inputLayerSize; j++) {
			tempSum += inputLayer[j] * weights[j * outputLayerSize + id]; // giri� n�ronlar� ve a��rl�klar �arp�l�p toplan�r.��k�� katman�na yaz�l�r
		} 
		outputLayer[id] = tempSum + BIAS * weights[inputLayerSize * outputLayerSize + id]; //bias de�eri eklenir

	}
}

__global__ void batchAndReLuDenseGPU(float* input, float* batchWeights, int inputSize) {
	int id = blockDim.x * blockIdx.x + threadIdx.x;  //bloklardaki threadlar belirlenip idleri tan�mlan�r

	if (id < inputSize) {
		float sDeviation = 0.0; 

		sDeviation = sqrt(batchWeights[(inputSize * 3) + id]);   //varyans kullan�larak standart sapma hesaplan�r. (featureCount * 3) dizide varyans elemanlar�na eri�ir
		input[id] = (input[id] - batchWeights[(inputSize * 2) + id]) / sDeviation;  //Her bir de�er aritmetik ortalamadan ��kar�l�p standart sapmaya b�l�n�r. (featureCount * 2) aritmetik ortalama de�erlerine eri�ir
		input[id] = input[id] * batchWeights[id] + batchWeights[inputSize + id];  //Sonu� gamma ile �arp�l�r beta ile toplan�r.

		if (fabs(input[id]) + input[id] < 0.00001) {
			input[id] = 0.0;
		}
	}
}


void model2Dense3ExecGPU(CpuGpuMem* cg)
{
	hipError_t result;

	int blockDim = 64;
	int threadCount = cg->denseOutputSize;
	int gridDim = (threadCount + blockDim - 1) / blockDim;

	int allocSize = threadCount * sizeof(float);

	result = hipFree(cg->gpuDensePtr);
	assert(result == hipSuccess);
	result = hipMalloc((float**)&cg->gpuDensePtr, allocSize);
	assert(result == hipSuccess);

	denseGPU << <gridDim, blockDim>> > (cg->gpuTempLayer2,cg->gpuDensePtr, cg->gpuDenseWeightPtr, cg->denseInputSize, cg->denseOutputSize);

	result = hipFree(cg->gpuTempLayer2);
	assert(result == hipSuccess);

	free(cg->cpuDensePtr);
	cg->cpuDensePtr = (float*)malloc(cg->denseOutputAllocSize);
	cpuGpuMemCopy(hipMemcpyDeviceToHost, cg, cg->cpuDensePtr, cg->gpuDensePtr, cg->denseOutputSize * sizeof(float));

}

void model2Dense2ExecGPU(CpuGpuMem* cg)
{
	hipError_t result;

	int blockDim = 64;
	int threadCount = cg->denseOutputSize;
	int gridDim = (threadCount + blockDim - 1) / blockDim;

	int allocSize = threadCount * sizeof(float);

	result = hipFree(cg->gpuTempLayer2);
	assert(result == hipSuccess);
	result = hipMalloc((float**)&cg->gpuTempLayer2, allocSize);
	assert(result == hipSuccess);

	denseGPU << <gridDim, blockDim>> > (cg->gpuDensePtr, cg->gpuTempLayer2, cg->gpuDenseWeightPtr, cg->denseInputSize, cg->denseOutputSize);
	
	batchAndReLuDenseGPU << <gridDim, blockDim>> > (cg->gpuTempLayer2, cg->gpuBatchPtr, cg->denseOutputSize);

}

void model2Dense1ExecGPU(CpuGpuMem* cg)
{
	hipError_t result;
	int mc = cg->maskCount;
	int fws = cg->featureWidthSize;
	int fhs = cg->featureHeightSize;


	int blockDim = 64;
	int threadCount = cg->maskCount * fws * fhs;
	int gridDim = (threadCount + blockDim - 1) / blockDim;


	int allocSize = threadCount * sizeof(float);

	result = hipMalloc((float**)&cg->gpuTempLayer2, allocSize);
	assert(result == hipSuccess);

	flattenGPU << <gridDim, blockDim>> > (cg->gpuTempLayer, cg->gpuTempLayer2, fws, fhs, mc);

	result = hipFree(cg->gpuTempLayer);
	assert(result == hipSuccess);

	threadCount = cg->denseOutputSize;
	gridDim = (threadCount + blockDim - 1) / blockDim;

	//clock_t tStart = clock();
	//double cpuClock = (double)(clock() - tStart) / CLOCKS_PER_SEC;
	denseGPU << <gridDim, blockDim>> > (cg->gpuTempLayer2, cg->gpuDensePtr, cg->gpuDenseWeightPtr, cg->denseInputSize, cg->denseOutputSize);

	batchAndReLuDenseGPU << <gridDim, blockDim>> > (cg->gpuDensePtr, cg->gpuBatchPtr, cg->denseOutputSize);
}

void model2Conv4ExecGpu(CpuGpuMem* cg)
{
	hipError_t result;
	int ms = cg->maskWHSize;
	int fws = cg->featureWidthSize;
	int fhs = cg->featureHeightSize;


	int blockDim = 64;
	int threadCount = cg->maskCount * (fws - ms + 1) * (fhs - ms + 1);
	int gridDim = (threadCount + blockDim - 1) / blockDim;


	int tempAllocSize = threadCount * sizeof(float);
	result = hipFree(cg->gpuTempLayer);
	assert(result == hipSuccess);
	result = hipMalloc((float**)&cg->gpuTempLayer, tempAllocSize);
	assert(result == hipSuccess);


	convHiddenGPU << <gridDim, blockDim >> > (cg->gpuFeaturePtr, cg->gpuTempLayer, cg->gpuMaskPtr, cg->featureWidthSize, cg->featureHeightSize, cg->maskWHSize, cg->maskCount, cg->maskDim);

	cg->featureWidthSize = fws - ms + 1;
	cg->featureHeightSize = fhs - ms + 1;

	batchNormGPU << <gridDim, blockDim >> > (cg->gpuTempLayer, cg->gpuBatchPtr, cg->featureWidthSize, cg->featureHeightSize, cg->maskCount);

	cpuGpuMemCopy(hipMemcpyDeviceToHost, cg, cg->cpuFeaturePtr, cg->gpuTempLayer, tempAllocSize);

}

void model2Conv3ExecGpu(CpuGpuMem* cg)
{
	hipError_t result;
	int ms = cg->maskWHSize;
	int fws = cg->featureWidthSize;
	int fhs = cg->featureHeightSize;


	int blockDim = 64;
	int threadCount = cg->maskCount * (fws - ms + 1) * (fhs - ms + 1);
	int gridDim = (threadCount + blockDim - 1) / blockDim;


	int tempAllocSize = threadCount * sizeof(float);
	result = hipFree(cg->gpuTempLayer);
	assert(result == hipSuccess);
	result = hipMalloc((float**)&cg->gpuTempLayer, tempAllocSize);
	assert(result == hipSuccess);


	convHiddenGPU << <gridDim, blockDim >> > (cg->gpuFeaturePtr, cg->gpuTempLayer, cg->gpuMaskPtr, cg->featureWidthSize, cg->featureHeightSize, cg->maskWHSize, cg->maskCount, cg->maskDim);

	cg->featureWidthSize = fws - ms + 1;
	cg->featureHeightSize = fhs - ms + 1;
	fws = cg->featureWidthSize;
	fhs = cg->featureHeightSize;

	batchNormGPU << <gridDim, blockDim>> > (cg->gpuTempLayer, cg->gpuBatchPtr, cg->featureWidthSize, cg->featureHeightSize, cg->maskCount);

	threadCount = cg->maskCount * (fws / cg->pool) * (fhs / cg->pool);
	gridDim = (threadCount + blockDim - 1) / blockDim;

	cg->featureAllocSize = threadCount * sizeof(float);
	free(cg->cpuFeaturePtr);
	cg->cpuFeaturePtr = (float*)malloc(cg->featureAllocSize);
	
	result = hipFree(cg->gpuFeaturePtr);
	assert(result == hipSuccess);
	result = hipMalloc((float**)&cg->gpuFeaturePtr, cg->featureAllocSize);
	assert(result == hipSuccess);

	maxPoolingGPU << <gridDim, blockDim >> > (cg->gpuTempLayer, cg->gpuFeaturePtr, cg->featureWidthSize, cg->featureHeightSize, cg->maskCount, cg->pool, cg->stride);

	cg->featureWidthSize /= cg->stride;
	cg->featureHeightSize /= cg->stride;

	cpuGpuMemCopy(hipMemcpyDeviceToHost, cg, cg->cpuFeaturePtr, cg->gpuFeaturePtr, cg->featureAllocSize);

}

void model2Conv2ExecGpu(CpuGpuMem* cg) {
	hipError_t result;
	int ms = cg->maskWHSize;
	int fws = cg->featureWidthSize;
	int fhs = cg->featureHeightSize;


	int blockDim = 64;
	int threadCount = cg->maskCount * (fws - ms + 1) * (fhs - ms + 1);
	int gridDim = (threadCount + blockDim - 1) / blockDim;


	cg->featureAllocSize = threadCount * sizeof(float);
	free(cg->cpuFeaturePtr);
	cg->cpuFeaturePtr = (float*)malloc(cg->featureAllocSize);
	result = hipFree(cg->gpuFeaturePtr);
	assert(result == hipSuccess);
	result = hipMalloc((float**)&cg->gpuFeaturePtr, cg->featureAllocSize);
	assert(result == hipSuccess);


	convHiddenGPU << <gridDim, blockDim >> > (cg->gpuTempLayer, cg->gpuFeaturePtr, cg->gpuMaskPtr, cg->featureWidthSize, cg->featureHeightSize, cg->maskWHSize, cg->maskCount, cg->maskDim);

	cg->featureWidthSize = fws - ms + 1;
	cg->featureHeightSize = fhs - ms + 1;

	batchNormGPU << <gridDim, blockDim>> > (cg->gpuFeaturePtr, cg->gpuBatchPtr, cg->featureWidthSize, cg->featureHeightSize, cg->maskCount);

	cpuGpuMemCopy(hipMemcpyDeviceToHost, cg, cg->cpuFeaturePtr, cg->gpuFeaturePtr, threadCount * sizeof(float));

}

void model2Conv1ExecGPU(CpuGpuMem* cg)
{
	hipError_t result;
	int iws = cg->imageWidthSize;
	int ihs = cg->imageHeightSize;
	int ms = cg->maskWHSize;
	int fws = cg->featureWidthSize = iws - ms + 1;
	int fhs = cg->featureHeightSize = ihs - ms + 1;

	int blockDim = 64;
	int threadCount = cg->maskCount * fws * fhs;


	int gridDim = (threadCount + blockDim - 1) / blockDim;


	conv1GPU << <gridDim, blockDim >> > ((int*)cg->gpuImagePtr, cg->gpuFeaturePtr, cg->gpuMaskPtr, cg->imageWidthSize, cg->imageHeightSize,
		cg->maskWHSize, cg->featureWidthSize, cg->featureHeightSize, cg->maskCount);

	batchNormGPU << <gridDim, blockDim>> > (cg->gpuFeaturePtr, cg->gpuBatchPtr, cg->featureWidthSize, cg->featureHeightSize, cg->maskCount);

	threadCount = cg->maskCount * (fws / cg->pool) * (fhs / cg->pool);
	gridDim = (threadCount + blockDim - 1) / blockDim;

	//temp array for pooling result
	result = hipMalloc((float**)&cg->gpuTempLayer, threadCount * sizeof(float));
	assert(result == hipSuccess);


	maxPoolingGPU << <gridDim, blockDim>> > (cg->gpuFeaturePtr, cg->gpuTempLayer, cg->featureWidthSize, cg->featureHeightSize, cg->maskCount, cg->pool, cg->stride);

	cg->featureWidthSize /= cg->stride;
	cg->featureHeightSize /= cg->stride;

	cpuGpuMemCopy(hipMemcpyDeviceToHost, cg, cg->cpuFeaturePtr, cg->gpuTempLayer, threadCount * sizeof(float));

}




void dense2ExecGPU(CpuGpuMem* cg)
{
	hipError_t result;

	int blockDim = 64;//Bir bloktaki thread say�s�
	int threadCount = cg->denseOutputSize;	//��k�� katman�ndaki toplam n�ron say�s� thread say�s�n� olu�turur. 
											//Bu durumda her n�ronun matematiksel hesab�n� o thread id sine sahip thread yapar.
	int gridDim = (threadCount + blockDim - 1) / blockDim; //gridDim toplam blok say�s�d�r. Bu normal threadCount / blockDim ile hesaplan�r. Fakat threadCount, blockDim'in tam katlar�ndan biri olmad���
														   //durumda do�ru blok say�s� olu�turulamaz. Bunun �n�ne ge�mek i�in (threadCount + blockDim - 1) / blockDim �eklinde hesaplanmal�

	int allocSize = threadCount * sizeof(float);

	result = hipFree(cg->gpuTempLayer2);// ��k�� n�ronlar�n�n sonu�lar�n�n yaz�laca�� bir ge�ici GPU bellek b�lgesi tahsis edilir.
	assert(result == hipSuccess);
	result = hipMalloc((float**)&cg->gpuTempLayer2, allocSize);
	assert(result == hipSuccess);

	denseGPU << <gridDim, blockDim >> > (cg->gpuDensePtr, cg->gpuTempLayer2, cg->gpuDenseWeightPtr, cg->denseInputSize, cg->denseOutputSize);//2. Tam ba�lant�l� katman�n GPU'da ko�mas�.

	free(cg->cpuDensePtr);//Sonu� RAM bellek b�lgesine aktar�lmadan �nce RAM pointer�n�n i�aret etti�i bellek b�lgesi serbest b�rak�l�r.
	cg->cpuDensePtr = (float*)malloc(cg->denseOutputAllocSize);
	cpuGpuMemCopy(hipMemcpyDeviceToHost, cg, cg->cpuDensePtr, cg->gpuTempLayer2, cg->denseOutputSize * sizeof(float)); //Hesaplamalar�n do�rulu�unu kontrol etmek ama�l� sonu�lar�n GPU belle�inden RAM belle�e transferi

}

void dense1ExecGPU(CpuGpuMem* cg)
{
	hipError_t result;
	int mc = cg->maskCount; //Maske say�s�
	int fws = cg->featureWidthSize; //Feature geni�li�i
	int fhs = cg->featureHeightSize; //Feature y�ksekli�i


	int blockDim = 64; //Bir bloktaki thread say�s�

	int threadCount = cg->maskCount * fws * fhs; //Flatten i�lemi i�in thread say�s� hesaplan�r.

	int gridDim = (threadCount + blockDim - 1) / blockDim; //gridDim toplam blok say�s�d�r. Bu normal threadCount / blockDim ile hesaplan�r. Fakat threadCount, blockDim'in tam katlar�ndan biri olmad���
														   //durumda do�ru blok say�s� olu�turulamaz. Bunun �n�ne ge�mek i�in (threadCount + blockDim - 1) / blockDim �eklinde hesaplanmal�


	int allocSize = threadCount * sizeof(float); // ��k�� n�ronlar�n�n sonu�lar�n�n yaz�laca�� bir ge�ici GPU bellek b�lgesi tahsis edilir.
	result = hipMalloc((float**)&cg->gpuTempLayer2, allocSize);
	assert(result == hipSuccess);

	flattenGPU << <gridDim, blockDim >> > (cg->gpuTempLayer, cg->gpuTempLayer2, fws, fhs, mc); //Feature uzay�ndaki feature'lar flatten i�lemine tabii tutulur.

	threadCount = cg->denseOutputSize; //��k�� katman�ndaki toplam n�ron say�s� thread say�s�n� olu�turur. 
									   //Bu durumda her n�ronun matematiksel hesab�n� o thread id sine sahip thread yapar.
	gridDim = (threadCount + blockDim - 1) / blockDim;

	//clock_t tStart = clock();
	//double cpuClock = (double)(clock() - tStart) / CLOCKS_PER_SEC;
	denseGPU << <gridDim, blockDim >> > (cg->gpuTempLayer2, cg->gpuDensePtr, cg->gpuDenseWeightPtr, cg->denseInputSize, cg->denseOutputSize); //1. Tam ba�lant�l� katman�n GPU'da ko�mas�.

	batchAndReLuDenseGPU << <gridDim, blockDim >> > (cg->gpuDensePtr, cg->gpuBatchPtr, cg->denseOutputSize);// 3. Batch Norm katman�n�n ve Relu i�leminin GPU'da ko�mas�

	cpuGpuMemCopy(hipMemcpyDeviceToHost, cg, cg->cpuDensePtr, cg->gpuDensePtr, cg->denseOutputSize * sizeof(float)); //Hesaplamalar�n do�rulu�unu kontrol etmek ama�l� sonu�lar�n GPU belle�inden RAM belle�e transferi
}

void convHidden1ExecGPU(CpuGpuMem* cg)
{
	hipError_t result;
	int ms = cg->maskWHSize;  //maske  boyutu (default=3)
	int fws = cg->featureWidthSize; //feature geni�li�i
	int fhs = cg->featureHeightSize; //feature y�ksekli�i


	int blockDim = 64; //Bir bloktaki thread say�s�
	int threadCount = cg->maskCount * (fws - ms + 1) * (fhs - ms + 1); //��k�� evri�im katman�ndaki toplam n�ron say�s� thread say�s�n� olu�uturur. 
																	   //Bu durumda her n�ronun matematiksel hesab�n� o thread id sine sahip thread yapar.

	int gridDim = (threadCount + blockDim - 1) / blockDim; //gridDim toplam blok say�s�d�r. Bu normal threadCount / blockDim ile hesaplan�r. Fakat threadCount, blockDim'in tam katlar�ndan biri olmad���
														   //durumda do�ru blok say�s� olu�turulamaz. Bunun �n�ne ge�mek i�in (threadCount + blockDim - 1) / blockDim �eklinde hesaplanmal�

	cg->featureAllocSize = threadCount * sizeof(float); //Gizli evri�im katman�n�n ��k���ndaki feature space i�in boyut g�ncellemesi yap�l�r. Ard�ndan RAM ve GPU bellek b�lgeleri tahsis edilir.
	free(cg->cpuFeaturePtr);
	cg->cpuFeaturePtr = (float*)malloc(cg->featureAllocSize);
	result = hipFree(cg->gpuFeaturePtr);
	assert(result == hipSuccess);
	result = hipMalloc((float**)&cg->gpuFeaturePtr, cg->featureAllocSize);
	assert(result == hipSuccess);
	hipMemset(cg->gpuFeaturePtr, 0, cg->featureAllocSize);//Gizli evri�im katman�n sonu�lar�n�n yaz�laca��, GPU bellek b�lgesinin i�eri�i s�f�ra e�itlenir.


	//Gizli evri�im katman�n�n GPU'da ko�mas�
	convHiddenGPU << <gridDim, blockDim >> > (cg->gpuTempLayer, cg->gpuFeaturePtr, cg->gpuMaskPtr, cg->featureWidthSize, cg->featureHeightSize, cg->maskWHSize, cg->maskCount, cg->maskDim);

	cg->featureWidthSize = fws - ms + 1;
	cg->featureHeightSize = fhs - ms + 1;
	fws = cg->featureWidthSize;
	fhs = cg->featureHeightSize;

	//2. Batch Norm katman�n�n GPU'da ko�mas�
	batchNormGPU << <gridDim, blockDim >> > (cg->gpuFeaturePtr, cg->gpuBatchPtr, cg->featureWidthSize, cg->featureHeightSize, cg->maskCount);

	threadCount = cg->maskCount * (fws / cg->pool) * (fhs / cg->pool); //Maxpool i�lemi i�in thread say�lar� belinlenmeli
	gridDim = (threadCount + blockDim - 1) / blockDim;

	result = hipFree(cg->gpuTempLayer);
	assert(result == hipSuccess);

	//Maxpool i�leminin sonucu i�in ge�ici bir bellek tahsisi yap�l�r.
	result = hipMalloc((float**)&cg->gpuTempLayer, threadCount * sizeof(float));
	assert(result == hipSuccess);

	maxPoolingGPU << <gridDim, blockDim >> > (cg->gpuFeaturePtr, cg->gpuTempLayer, cg->featureWidthSize, cg->featureHeightSize, cg->maskCount, cg->pool, cg->stride);

	cg->featureWidthSize /= cg->stride;// maxpool sonunda feature boyutu g�ncellenir
	cg->featureHeightSize /= cg->stride;

	cpuGpuMemCopy(hipMemcpyDeviceToHost, cg, cg->cpuFeaturePtr, cg->gpuTempLayer, threadCount * sizeof(float)); // Gizli Evri�im katman�n�n sonucu GPU bellekten RAM belle�e transfer edilir.

}

void conv1ExecGPU(CpuGpuMem* cg)
{
	hipError_t result;
	int iws = cg->imageWidthSize; //g�r�nt� geni�li�i
	int ihs = cg->imageHeightSize; //g�r�nt� y�ksekli�i
	int ms = cg->maskWHSize;  //maske  boyutu (default=3)
	int fws = cg->featureWidthSize = iws - ms + 1; //feature geni�ili�i
	int fhs = cg->featureHeightSize = ihs - ms + 1;//feature y�ksekli�i

	int blockDim = 64; //Bir bloktaki thread say�s�
	int threadCount = cg->maskCount * fws * fhs; //��k�� evri�im katman�ndaki toplam n�ron say�s� thread say�s�n� olu�uturur. 
												 //Bu durumda her n�ronun matematiksel hesab�n� o thread id sine sahip thread yapar.

	int gridDim = (threadCount + blockDim - 1) / blockDim; //gridDim toplam blok say�s�d�r. Bu normal threadCount / blockDim ile hesaplan�r. Fakat threadCount, blockDim'in tam katlar�ndan biri olmad���
														   //durumda do�ru blok say�s� olu�turulamaz. Bunun �n�ne ge�mek i�in (threadCount + blockDim - 1) / blockDim �eklinde hesaplanmal�

	conv1GPU << <gridDim, blockDim >> > ((int*)cg->gpuImagePtr, cg->gpuFeaturePtr, cg->gpuMaskPtr, cg->imageWidthSize, cg->imageHeightSize,
		cg->maskWHSize, cg->featureWidthSize, cg->featureHeightSize, cg->maskCount); //1. evri�im katman� GPU'da ko�mas�

	batchNormGPU << <gridDim, blockDim >> > (cg->gpuFeaturePtr, cg->gpuBatchPtr, cg->featureWidthSize, cg->featureHeightSize, cg->maskCount); //1. batch norm GPU'da ko�mas�

	threadCount = cg->maskCount * (fws / cg->pool) * (fhs / cg->pool); //Maxpool i�lemi i�in thread say�lar� belinlenmeli
	gridDim = (threadCount + blockDim - 1) / blockDim;

	//Maxpool i�leminin sonucu i�in ge�ici bir bellek tahsisi yap�l�r
	result = hipMalloc((float**)&cg->gpuTempLayer, threadCount * sizeof(float));
	assert(result == hipSuccess);


	maxPoolingGPU << <gridDim, blockDim >> > (cg->gpuFeaturePtr, cg->gpuTempLayer, cg->featureWidthSize, cg->featureHeightSize, cg->maskCount, cg->pool, cg->stride); //maxpool GPU'da ko�mas�

	cg->featureWidthSize /= cg->stride; // maxpool sonunda feature boyutu g�ncellenir
	cg->featureHeightSize /= cg->stride;

	cpuGpuMemCopy(hipMemcpyDeviceToHost, cg, cg->cpuFeaturePtr, cg->gpuTempLayer, threadCount * sizeof(float)); // 1. katman�n sonucu GPU bellekten RAM belle�e transfer edilir.

}


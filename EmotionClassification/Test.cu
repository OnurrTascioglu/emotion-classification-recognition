#include "hip/hip_runtime.h"
#include <windows.h>
#include <cmath>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>
#include ""

#include "CpuGpu.h"
#include "CpuGpuMem.h"
#include "KernelGpuAdd.cuh"
#include "hip/hip_runtime.h"

#define BIAS 1

//----------------conv1
__global__ void conv1GPU(float* resultImages, float* masks, int* image, int width, int height, int maskSize, int rMatrixWidth, int rMatrixHeight, int maskCount)
{
	int id = blockDim.x * blockIdx.x + threadIdx.x;

	if (id < maskCount * rMatrixWidth * rMatrixHeight) {
		int temp = 0;
		int j = id % rMatrixWidth;
		temp = id / rMatrixWidth;
		int i = temp % rMatrixHeight;
		int m = temp / rMatrixHeight;

		for (int k = 0; k < maskSize * maskSize; k++) {
			int mCol = k % maskSize;
			int mRow = k / maskSize;
			resultImages[(m * rMatrixWidth * rMatrixHeight) + i * rMatrixWidth + j] +=
				(float)image[(width * i + j) + mRow * width + mCol] * masks[m * (maskSize * maskSize) + k];
		}
		resultImages[(m * rMatrixWidth * rMatrixHeight) + i * rMatrixWidth + j] += (float)BIAS * masks[maskCount * (maskSize * maskSize) + m];
	}
}

__global__ void batchNormGPU(float* feature, float* batchWeights, int width, int height, int featureCount)
{
	int id = blockDim.x * blockIdx.x + threadIdx.x;

	if (id < featureCount * width * height) {
		int i = id % (width * height);
		int m = id / (width * height);

		float sDeviation = 0.0; // standart sapma i�in

		sDeviation = sqrt(batchWeights[(featureCount * 3) + m]);

		feature[(m * width * height) + i] = (feature[(m * width * height) + i] - batchWeights[featureCount * 2 + m]) / sDeviation;
		feature[(m * width * height) + i] = feature[(m * width * height) + i] * batchWeights[m] + batchWeights[featureCount + m];

		if (fabs(feature[(m * width * height) + i]) + feature[(m * width * height) + i] < 0.0001) {
			feature[(m * width * height) + i] = 0.0;
		}
	}
}

__global__ void maxPoolingGPU(float* feature, float* tempFeature, int width, int height, int  featureCount, int pool, int stride)
{
	int id = blockDim.x * blockIdx.x + threadIdx.x;


	if (id < featureCount * (width / stride) * (height / stride)) {
		int temp2 = 0;
		int col = id % (width / stride);
		temp2 = id / (width / stride);
		int row = temp2 % (height / stride);
		int m = temp2 / (height / stride);

		float max = 0.0;
		float temp = 0.0;



		for (int k = 0; k < pool; k++) {
			for (int n = 0; n < pool; n++) {
				temp = feature[(m * width * height) + row * width * stride + col * stride + k * width + n];
				if ((temp - max) > 0.0001) {
					max = temp;
				}
			}
		}
		tempFeature[(m * (width / stride) * (height / stride)) + (row * (width / stride)) + col] = max;

	}

}

//-----------------------conv2


__global__ void convHiddenGPU(float* resultImages, float* feature, float* weights, int fWidth, int fHeight, int maskSize, int maskCount, int maskDim)
{
	int id = blockDim.x * blockIdx.x + threadIdx.x;

	int rMatrixWidth = fWidth - maskSize + 1;
	int rMatrixHeight = fHeight - maskSize + 1;

	if (id < maskCount * rMatrixWidth * rMatrixHeight) {
		int temp = 0;
		int j = id % rMatrixWidth;
		temp = id / rMatrixWidth;
		int i = temp % rMatrixHeight;
		int c = temp / rMatrixHeight;

		for (int d = 0; d < maskDim; d++) {
			for (int k = 0; k < maskSize * maskSize; k++) {
				int mCol = k % maskSize;
				int mRow = k / maskSize;
				resultImages[(c * rMatrixWidth * rMatrixHeight) + i * rMatrixWidth + j] +=
					(float)feature[d * fWidth * fHeight + (fWidth * i + j) + mRow * fWidth + mCol] * weights[c * (maskDim * maskSize * maskSize) + d * maskSize * maskSize + k];

			}
		}
		resultImages[(c * rMatrixWidth * rMatrixHeight) + i * rMatrixWidth + j] += BIAS * weights[maskCount * maskDim * maskSize * maskSize + c];
	}
}



void convHidden1ExecGPU(CpuGpuMem* cg)
{
	hipError_t result;
	int ms = cg->maskWHSize;
	int fws = cg->featureWidthSize;
	int fhs = cg->featureHeightSize;
	int dfhs = cg->dtoFeatureHeightSize;
	int dfws = cg->dtoFeatureWidthSize;


	int blockDim = 1024;
	int threadCount = cg->maskCount * (fws - ms + 1) * (fhs - ms + 1);
	int gridDim = (threadCount + blockDim - 1) / blockDim;


	convHiddenGPU << <gridDim, blockDim, 0, cg->stream >> > (cg->gpuDtoFeaturePtr, cg->gpuFeaturePtrTemp, cg->gpuMaskPtr, cg->featureWidthSize, cg->featureHeightSize, cg->maskWHSize, cg->maskCount, cg->maskDim);

	cg->featureWidthSize = fws - ms + 1;
	cg->featureHeightSize = fhs - ms + 1;

	batchNormGPU << <gridDim, blockDim, 0, cg->stream >> > (cg->gpuDtoFeaturePtr, cg->gpuBatchPtr, cg->dtoFeatureWidthSize, cg->dtoFeatureHeightSize, cg->maskCount);


	threadCount = cg->maskCount * (fws / cg->pool) * (fhs / cg->pool);
	gridDim = (threadCount + blockDim - 1) / blockDim;

	result = hipFree(cg->gpuFeaturePtrTemp);
	assert(result == hipSuccess);
	result = hipMalloc((float**)&cg->gpuFeaturePtrTemp, threadCount * sizeof(float));
	assert(result == hipSuccess);

	maxPoolingGPU << <gridDim, blockDim, 0, cg->stream >> > (cg->gpuDtoFeaturePtr, cg->gpuFeaturePtrTemp, cg->dtoFeatureWidthSize, cg->dtoFeatureHeightSize, cg->maskCount, cg->pool, cg->stride);
	cpuGpuMemCopy(hipMemcpyDeviceToHost, cg, cg->cpuDtoFeaturePtr, cg->gpuFeaturePtrTemp, threadCount * sizeof(float));

	cg->featureWidthSize /= cg->stride;
	cg->featureHeightSize /= cg->stride;

}


void conv1ExecGPU(CpuGpuMem* cg)
{
	hipError_t result;
	int iws = cg->imageWidthSize;
	int ihs = cg->imageHeightSize;
	int ms = cg->maskWHSize;
	int fws = cg->featureWidthSize = iws - ms + 1;
	int fhs = cg->featureHeightSize = ihs - ms + 1;

	int blockDim = 1024;
	int threadCount = cg->maskCount * fws * fhs;


	int gridDim = (threadCount + blockDim - 1) / blockDim;


	conv1GPU << <gridDim, blockDim, 0, cg->stream >> > (cg->gpuFeaturePtr, cg->gpuMaskPtr, (int*)cg->gpuImagePtr, cg->imageWidthSize, cg->imageHeightSize,
		cg->maskWHSize, cg->featureWidthSize, cg->featureHeightSize, cg->maskCount);

	batchNormGPU << <gridDim, blockDim, 0, cg->stream >> > (cg->gpuFeaturePtr, cg->gpuBatchPtr, cg->featureWidthSize, cg->featureHeightSize, cg->maskCount);

	threadCount = cg->maskCount * (fws / cg->pool) * (fhs / cg->pool);
	gridDim = (threadCount + blockDim - 1) / blockDim;

	//temp array for pooling result
	result = hipMalloc((float**)&cg->gpuFeaturePtrTemp, threadCount * sizeof(float));
	assert(result == hipSuccess);

	maxPoolingGPU << <gridDim, blockDim, 0, cg->stream >> > (cg->gpuFeaturePtr, cg->gpuFeaturePtrTemp, cg->featureWidthSize, cg->featureHeightSize, cg->maskCount, cg->pool, cg->stride);

	cg->featureWidthSize /= cg->stride;
	cg->featureHeightSize /= cg->stride;

}




#include "hip/hip_runtime.h"
#include <windows.h>
#include <cmath>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>
#include ""
#include <time.h>
#include "CpuGpu.h"
#include "CpuGpuMem.h"
#include "KernelGpuAdd.cuh"
#include "hip/hip_runtime.h"

#define BIAS 1

//----------------conv1
__global__ void conv1GPU(int* image, float* resultImages, float* masks, int width, int height, int maskSize, int rMatrixWidth, int rMatrixHeight, int maskCount)
{
	int id = blockDim.x * blockIdx.x + threadIdx.x;

	if (id < maskCount * rMatrixWidth * rMatrixHeight) {
		int temp = 0;
		int j = id % rMatrixWidth;
		temp = id / rMatrixWidth;
		int i = temp % rMatrixHeight;
		int m = temp / rMatrixHeight;
		float tempSum = 0.0;

		for (int k = 0; k < maskSize * maskSize; k++) {
			int mCol = k % maskSize;
			int mRow = k / maskSize;
			tempSum +=
				(float)image[(width * i + j) + mRow * width + mCol] * masks[m * (maskSize * maskSize) + k];
		}
		resultImages[(m * rMatrixWidth * rMatrixHeight) + i * rMatrixWidth + j] =  tempSum + (float)BIAS * masks[maskCount * (maskSize * maskSize) + m];
	}
}

__global__ void batchNormGPU(float* feature, float* batchWeights, int width, int height, int featureCount)
{
	int id = blockDim.x * blockIdx.x + threadIdx.x;

	if (id < featureCount * width * height) {
		int i = id % (width * height);
		int m = id / (width * height);

		float sDeviation = 0.0; // standart sapma i�in

		sDeviation = sqrt(batchWeights[(featureCount * 3) + m]);

		feature[(m * width * height) + i] = (feature[(m * width * height) + i] - batchWeights[featureCount * 2 + m]) / sDeviation;
		feature[(m * width * height) + i] = feature[(m * width * height) + i] * batchWeights[m] + batchWeights[featureCount + m];

		if (fabs(feature[(m * width * height) + i]) + feature[(m * width * height) + i] < 0.001) {
			feature[(m * width * height) + i] = 0.0;
		}
	}
}

__global__ void maxPoolingGPU(float* feature, float* tempFeature, int width, int height, int  featureCount, int pool, int stride)
{
	int id = blockDim.x * blockIdx.x + threadIdx.x;


	if (id < featureCount * (width / stride) * (height / stride)) {
		int temp2 = 0;
		int col = id % (width / stride);
		temp2 = id / (width / stride);
		int row = temp2 % (height / stride);
		int m = temp2 / (height / stride);

		float max = 0.0;
		float temp = 0.0;



		for (int k = 0; k < pool; k++) {
			for (int n = 0; n < pool; n++) {
				temp = feature[(m * width * height) + row * width * stride + col * stride + k * width + n];
				if ((temp - max) > 0.00001) {
					max = temp;
				}
			}
		}
		tempFeature[(m * (width / stride) * (height / stride)) + (row * (width / stride)) + col] = max;

	}

}

//-----------------------conv2


__global__ void convHiddenGPU(float* feature, float* resultImages, float* weights, int fWidth, int fHeight, int maskSize, int maskCount, int maskDim)
{
	int id = blockDim.x * blockIdx.x + threadIdx.x;

	int rMatrixWidth = fWidth - maskSize + 1;
	int rMatrixHeight = fHeight - maskSize + 1;

	if (id < maskCount * rMatrixWidth * rMatrixHeight) {
		int temp = 0;
		int j = id % rMatrixWidth;
		temp = id / rMatrixWidth;
		int i = temp % rMatrixHeight;
		int c = temp / rMatrixHeight;
		float tempSum = 0.0;

		for (int d = 0; d < maskDim; d++) {
			for (int k = 0; k < maskSize * maskSize; k++) {
				int mCol = k % maskSize;
				int mRow = k / maskSize;
				tempSum +=	(float)feature[d * fWidth * fHeight + (fWidth * i + j) + mRow * fWidth + mCol] * weights[c * (maskDim * maskSize * maskSize) + d * maskSize * maskSize + k];

			}
		}
		resultImages[(c * rMatrixWidth * rMatrixHeight) + i * rMatrixWidth + j] = tempSum + BIAS * weights[maskCount * maskDim * maskSize * maskSize + c];
	}
}

__global__ void flattenGPU(float* features, float* flattenArray, int width, int height, int featureCount) {
	int id = blockDim.x * blockIdx.x + threadIdx.x;

	if (id < featureCount * width * height) {

		int temp = 0;
		int f = id % featureCount;
		temp = id / featureCount;
		int j = temp % width;
		int i = temp / width;

		flattenArray[id] = features[f * width * height + i * width + j];
	}
}

__global__ void denseGPU(float* inputLayer, float* outputLayer, float* weights, int inputLayerSize, int outputLayerSize) {
	int id = blockDim.x * blockIdx.x + threadIdx.x;

	if (id < outputLayerSize) {
		// optimize edilmeli
		float tempSum = 0.0;

		for (int j = 0; j < inputLayerSize; j++) {
			tempSum += inputLayer[j] * weights[j * outputLayerSize + id];
		}
		outputLayer[id] = tempSum + BIAS * weights[inputLayerSize * outputLayerSize + id];

	}
}

__global__ void batchAndReLuDenseGPU(float* input, float* batchWeights, int inputSize) {
	int id = blockDim.x * blockIdx.x + threadIdx.x;

	if (id < inputSize) {
		float sDeviation = 0.0; 

		sDeviation = sqrt(batchWeights[(inputSize * 3) + id]);
		input[id] = (input[id] - batchWeights[(inputSize * 2) + id]) / sDeviation;
		input[id] = input[id] * batchWeights[id] + batchWeights[inputSize + id];

		if (fabs(input[id]) + input[id] < 0.00001) {
			input[id] = 0.0;
		}
	}
}


void model2Dense3ExecGPU(CpuGpuMem* cg)
{
	hipError_t result;

	int blockDim = 64;
	int threadCount = cg->denseOutputSize;
	int gridDim = (threadCount + blockDim - 1) / blockDim;

	int allocSize = threadCount * sizeof(float);

	result = hipFree(cg->gpuDensePtr);
	assert(result == hipSuccess);
	result = hipMalloc((float**)&cg->gpuDensePtr, allocSize);
	assert(result == hipSuccess);

	hipMemset(cg->gpuTempLayer2, 0, allocSize);
	denseGPU << <gridDim, blockDim, 0, cg->stream >> > (cg->gpuTempLayer2,cg->gpuDensePtr, cg->gpuDenseWeightPtr, cg->denseInputSize, cg->denseOutputSize);

	free(cg->cpuDensePtr);
	cg->cpuDensePtr = (float*)malloc(cg->denseOutputAllocSize);
	cpuGpuMemCopy(hipMemcpyDeviceToHost, cg, cg->cpuDensePtr, cg->gpuDensePtr, cg->denseOutputSize * sizeof(float));

}

void model2Dense2ExecGPU(CpuGpuMem* cg)
{
	hipError_t result;

	int blockDim = 64;
	int threadCount = cg->denseOutputSize;
	int gridDim = (threadCount + blockDim - 1) / blockDim;

	int allocSize = threadCount * sizeof(float);

	result = hipFree(cg->gpuTempLayer2);
	assert(result == hipSuccess);
	result = hipMalloc((float**)&cg->gpuTempLayer2, allocSize);
	assert(result == hipSuccess);

	denseGPU << <gridDim, blockDim, 0, cg->stream >> > (cg->gpuDensePtr, cg->gpuTempLayer2, cg->gpuDenseWeightPtr, cg->denseInputSize, cg->denseOutputSize);
	
	batchAndReLuDenseGPU << <gridDim, blockDim, 0, cg->stream >> > (cg->gpuTempLayer2, cg->gpuBatchPtr, cg->denseOutputSize);

	//free(cg->cpuDensePtr);
	//cg->cpuDensePtr = (float*)malloc(cg->denseOutputAllocSize);

	//cpuGpuMemCopy(hipMemcpyDeviceToHost, cg, cg->cpuDensePtr, cg->gpuTempLayer2, cg->denseOutputSize * sizeof(float));

}

void model2Dense1ExecGPU(CpuGpuMem* cg)
{
	hipError_t result;
	int mc = cg->maskCount;
	int fws = cg->featureWidthSize;
	int fhs = cg->featureHeightSize;


	int blockDim = 64;
	int threadCount = cg->maskCount * fws * fhs;
	int gridDim = (threadCount + blockDim - 1) / blockDim;


	int allocSize = threadCount * sizeof(float);

	result = hipMalloc((float**)&cg->gpuTempLayer2, allocSize);
	assert(result == hipSuccess);

	flattenGPU << <gridDim, blockDim, 0, cg->stream >> > (cg->gpuTempLayer, cg->gpuTempLayer2, fws, fhs, mc);

	threadCount = cg->denseOutputSize;
	gridDim = (threadCount + blockDim - 1) / blockDim;

	//clock_t tStart = clock();
	//double cpuClock = (double)(clock() - tStart) / CLOCKS_PER_SEC;
	denseGPU << <gridDim, blockDim, 0, cg->stream >> > (cg->gpuTempLayer2, cg->gpuDensePtr, cg->gpuDenseWeightPtr, cg->denseInputSize, cg->denseOutputSize);

	batchAndReLuDenseGPU << <gridDim, blockDim, 0, cg->stream >> > (cg->gpuDensePtr, cg->gpuBatchPtr, cg->denseOutputSize);

}

void model2Conv4ExecGpu(CpuGpuMem* cg)
{
	hipError_t result;
	int ms = cg->maskWHSize;
	int fws = cg->featureWidthSize;
	int fhs = cg->featureHeightSize;


	int blockDim = 64;
	int threadCount = cg->maskCount * (fws - ms + 1) * (fhs - ms + 1);
	int gridDim = (threadCount + blockDim - 1) / blockDim;


	int tempAllocSize = threadCount * sizeof(float);
	result = hipFree(cg->gpuTempLayer);
	assert(result == hipSuccess);
	result = hipMalloc((float**)&cg->gpuTempLayer, tempAllocSize);
	assert(result == hipSuccess);


	convHiddenGPU << <gridDim, blockDim, 0, cg->stream >> > (cg->gpuFeaturePtr, cg->gpuTempLayer, cg->gpuMaskPtr, cg->featureWidthSize, cg->featureHeightSize, cg->maskWHSize, cg->maskCount, cg->maskDim);

	cg->featureWidthSize = fws - ms + 1;
	cg->featureHeightSize = fhs - ms + 1;

	batchNormGPU << <gridDim, blockDim, 0, cg->stream >> > (cg->gpuTempLayer, cg->gpuBatchPtr, cg->featureWidthSize, cg->featureHeightSize, cg->maskCount);

	cpuGpuMemCopy(hipMemcpyDeviceToHost, cg, cg->cpuFeaturePtr, cg->gpuTempLayer, tempAllocSize);

}

void model2Conv3ExecGpu(CpuGpuMem* cg)
{
	hipError_t result;
	int ms = cg->maskWHSize;
	int fws = cg->featureWidthSize;
	int fhs = cg->featureHeightSize;


	int blockDim = 64;
	int threadCount = cg->maskCount * (fws - ms + 1) * (fhs - ms + 1);
	int gridDim = (threadCount + blockDim - 1) / blockDim;


	int tempAllocSize = threadCount * sizeof(float);
	result = hipFree(cg->gpuTempLayer);
	assert(result == hipSuccess);
	result = hipMalloc((float**)&cg->gpuTempLayer, tempAllocSize);
	assert(result == hipSuccess);


	convHiddenGPU << <gridDim, blockDim, 0, cg->stream >> > (cg->gpuFeaturePtr, cg->gpuTempLayer, cg->gpuMaskPtr, cg->featureWidthSize, cg->featureHeightSize, cg->maskWHSize, cg->maskCount, cg->maskDim);

	cg->featureWidthSize = fws - ms + 1;
	cg->featureHeightSize = fhs - ms + 1;

	batchNormGPU << <gridDim, blockDim, 0, cg->stream >> > (cg->gpuTempLayer, cg->gpuBatchPtr, cg->featureWidthSize, cg->featureHeightSize, cg->maskCount);

	threadCount = cg->maskCount * (fws / cg->pool) * (fhs / cg->pool);
	gridDim = (threadCount + blockDim - 1) / blockDim;

	cg->featureAllocSize = threadCount * sizeof(float);
	free(cg->cpuFeaturePtr);
	cg->cpuFeaturePtr = (float*)malloc(cg->featureAllocSize);
	result = hipFree(cg->gpuFeaturePtr);
	assert(result == hipSuccess);

	result = hipMalloc((float**)&cg->gpuFeaturePtr, cg->featureAllocSize);
	assert(result == hipSuccess);

	maxPoolingGPU << <gridDim, blockDim, 0, cg->stream >> > (cg->gpuTempLayer, cg->gpuFeaturePtr, cg->featureWidthSize, cg->featureHeightSize, cg->maskCount, cg->pool, cg->stride);

	cg->featureWidthSize /= cg->stride;
	cg->featureHeightSize /= cg->stride;

	cpuGpuMemCopy(hipMemcpyDeviceToHost, cg, cg->cpuFeaturePtr, cg->gpuFeaturePtr, cg->featureAllocSize);

}

void model2Conv2ExecGpu(CpuGpuMem* cg) {
	hipError_t result;
	int ms = cg->maskWHSize;
	int fws = cg->featureWidthSize;
	int fhs = cg->featureHeightSize;


	int blockDim = 64;
	int threadCount = cg->maskCount * (fws - ms + 1) * (fhs - ms + 1);
	int gridDim = (threadCount + blockDim - 1) / blockDim;


	cg->featureAllocSize = threadCount * sizeof(float);
	free(cg->cpuFeaturePtr);
	cg->cpuFeaturePtr = (float*)malloc(cg->featureAllocSize);
	result = hipFree(cg->gpuFeaturePtr);
	assert(result == hipSuccess);
	result = hipMalloc((float**)&cg->gpuFeaturePtr, cg->featureAllocSize);
	assert(result == hipSuccess);


	convHiddenGPU << <gridDim, blockDim, 0, cg->stream >> > (cg->gpuTempLayer, cg->gpuFeaturePtr, cg->gpuMaskPtr, cg->featureWidthSize, cg->featureHeightSize, cg->maskWHSize, cg->maskCount, cg->maskDim);

	cg->featureWidthSize = fws - ms + 1;
	cg->featureHeightSize = fhs - ms + 1;

	batchNormGPU << <gridDim, blockDim, 0, cg->stream >> > (cg->gpuFeaturePtr, cg->gpuBatchPtr, cg->featureWidthSize, cg->featureHeightSize, cg->maskCount);

	cpuGpuMemCopy(hipMemcpyDeviceToHost, cg, cg->cpuFeaturePtr, cg->gpuFeaturePtr, threadCount * sizeof(float));

}

void model2Conv1ExecGPU(CpuGpuMem* cg)
{
	hipError_t result;
	int iws = cg->imageWidthSize;
	int ihs = cg->imageHeightSize;
	int ms = cg->maskWHSize;
	int fws = cg->featureWidthSize = iws - ms + 1;
	int fhs = cg->featureHeightSize = ihs - ms + 1;

	int blockDim = 64;
	int threadCount = cg->maskCount * fws * fhs;


	int gridDim = (threadCount + blockDim - 1) / blockDim;


	conv1GPU << <gridDim, blockDim, 0, cg->stream >> > ((int*)cg->gpuImagePtr, cg->gpuFeaturePtr, cg->gpuMaskPtr, cg->imageWidthSize, cg->imageHeightSize,
		cg->maskWHSize, cg->featureWidthSize, cg->featureHeightSize, cg->maskCount);

	batchNormGPU << <gridDim, blockDim, 0, cg->stream >> > (cg->gpuFeaturePtr, cg->gpuBatchPtr, cg->featureWidthSize, cg->featureHeightSize, cg->maskCount);

	threadCount = cg->maskCount * (fws / cg->pool) * (fhs / cg->pool);
	gridDim = (threadCount + blockDim - 1) / blockDim;

	//temp array for pooling result
	result = hipMalloc((float**)&cg->gpuTempLayer, threadCount * sizeof(float));
	assert(result == hipSuccess);


	maxPoolingGPU << <gridDim, blockDim, 0, cg->stream >> > (cg->gpuFeaturePtr, cg->gpuTempLayer, cg->featureWidthSize, cg->featureHeightSize, cg->maskCount, cg->pool, cg->stride);

	cg->featureWidthSize /= cg->stride;
	cg->featureHeightSize /= cg->stride;

	cpuGpuMemCopy(hipMemcpyDeviceToHost, cg, cg->cpuFeaturePtr, cg->gpuTempLayer, threadCount * sizeof(float));

}



void dense2ExecGPU(CpuGpuMem* cg)
{
	hipError_t result;

	int blockDim = 64;
	int threadCount = cg->denseOutputSize;
	int gridDim = (threadCount + blockDim - 1) / blockDim;

	int allocSize = threadCount * sizeof(float);

	result = hipFree(cg->gpuTempLayer2);
	assert(result == hipSuccess);
	result = hipMalloc((float**)&cg->gpuTempLayer2, allocSize);
	assert(result == hipSuccess);

	hipMemset(cg->gpuTempLayer2, 0, allocSize);
	denseGPU << <gridDim, blockDim, 0, cg->stream >> > (cg->gpuDensePtr, cg->gpuTempLayer2, cg->gpuDenseWeightPtr, cg->denseInputSize, cg->denseOutputSize);

	free(cg->cpuDensePtr);
	cg->cpuDensePtr = (float*)malloc(cg->denseOutputAllocSize);
	cpuGpuMemCopy(hipMemcpyDeviceToHost, cg, cg->cpuDensePtr, cg->gpuTempLayer2, cg->denseOutputSize * sizeof(float));

}

void dense1ExecGPU(CpuGpuMem* cg)
{
	hipError_t result;
	int mc = cg->maskCount;
	int fws = cg->featureWidthSize;
	int fhs = cg->featureHeightSize;


	int blockDim = 64;
	int threadCount = cg->maskCount * fws * fhs;
	int gridDim = (threadCount + blockDim - 1) / blockDim;


	int allocSize = threadCount * sizeof(float);

	result = hipMalloc((float**)&cg->gpuTempLayer2, allocSize);
	assert(result == hipSuccess);

	flattenGPU << <gridDim, blockDim, 0, cg->stream >> > (cg->gpuTempLayer, cg->gpuTempLayer2, fws, fhs, mc);

	threadCount = cg->denseOutputSize;
	gridDim = (threadCount + blockDim - 1) / blockDim;

	//clock_t tStart = clock();
	//double cpuClock = (double)(clock() - tStart) / CLOCKS_PER_SEC;
	denseGPU << <gridDim, blockDim, 0, cg->stream >> > (cg->gpuTempLayer2, cg->gpuDensePtr, cg->gpuDenseWeightPtr, cg->denseInputSize, cg->denseOutputSize);

	batchAndReLuDenseGPU << <gridDim, blockDim, 0, cg->stream >> > (cg->gpuDensePtr, cg->gpuBatchPtr, cg->denseOutputSize);

	cpuGpuMemCopy(hipMemcpyDeviceToHost, cg, cg->cpuDensePtr, cg->gpuDensePtr, cg->denseOutputSize * sizeof(float));
}

void convHidden1ExecGPU(CpuGpuMem* cg)
{
	hipError_t result;
	int ms = cg->maskWHSize;
	int fws = cg->featureWidthSize;
	int fhs = cg->featureHeightSize;


	int blockDim = 64;
	int threadCount = cg->maskCount * (fws - ms + 1) * (fhs - ms + 1);
	int gridDim = (threadCount + blockDim - 1) / blockDim;


	cg->featureAllocSize = threadCount * sizeof(float);
	free(cg->cpuFeaturePtr);
	cg->cpuFeaturePtr = (float*)malloc(cg->featureAllocSize);
	result = hipFree(cg->gpuFeaturePtr);
	assert(result == hipSuccess);
	result = hipMalloc((float**)&cg->gpuFeaturePtr, cg->featureAllocSize);
	assert(result == hipSuccess);
	hipMemset(cg->gpuFeaturePtr, 0, cg->featureAllocSize);


	convHiddenGPU << <gridDim, blockDim, 0, cg->stream >> > (cg->gpuTempLayer, cg->gpuFeaturePtr, cg->gpuMaskPtr, cg->featureWidthSize, cg->featureHeightSize, cg->maskWHSize, cg->maskCount, cg->maskDim);

	cg->featureWidthSize = fws - ms + 1;
	cg->featureHeightSize = fhs - ms + 1;

	batchNormGPU << <gridDim, blockDim, 0, cg->stream >> > (cg->gpuFeaturePtr, cg->gpuBatchPtr, cg->featureWidthSize, cg->featureHeightSize, cg->maskCount);

	threadCount = cg->maskCount * (fws / cg->pool) * (fhs / cg->pool);
	gridDim = (threadCount + blockDim - 1) / blockDim;

	result = hipFree(cg->gpuTempLayer);
	assert(result == hipSuccess);

	result = hipMalloc((float**)&cg->gpuTempLayer, threadCount * sizeof(float));
	assert(result == hipSuccess);

	maxPoolingGPU << <gridDim, blockDim, 0, cg->stream >> > (cg->gpuFeaturePtr, cg->gpuTempLayer, cg->featureWidthSize, cg->featureHeightSize, cg->maskCount, cg->pool, cg->stride);

	cg->featureWidthSize /= cg->stride;
	cg->featureHeightSize /= cg->stride;

	cpuGpuMemCopy(hipMemcpyDeviceToHost, cg, cg->cpuFeaturePtr, cg->gpuTempLayer, threadCount * sizeof(float));

}

void conv1ExecGPU(CpuGpuMem* cg)
{
	hipError_t result;
	int iws = cg->imageWidthSize;
	int ihs = cg->imageHeightSize;
	int ms = cg->maskWHSize;
	int fws = cg->featureWidthSize = iws - ms + 1;
	int fhs = cg->featureHeightSize = ihs - ms + 1;

	int blockDim = 64;
	int threadCount = cg->maskCount * fws * fhs;


	int gridDim = (threadCount + blockDim - 1) / blockDim;


	conv1GPU << <gridDim, blockDim, 0, cg->stream >> > ((int*)cg->gpuImagePtr, cg->gpuFeaturePtr, cg->gpuMaskPtr, cg->imageWidthSize, cg->imageHeightSize,
		cg->maskWHSize, cg->featureWidthSize, cg->featureHeightSize, cg->maskCount);

	batchNormGPU << <gridDim, blockDim, 0, cg->stream >> > (cg->gpuFeaturePtr, cg->gpuBatchPtr, cg->featureWidthSize, cg->featureHeightSize, cg->maskCount);

	threadCount = cg->maskCount * (fws / cg->pool) * (fhs / cg->pool);
	gridDim = (threadCount + blockDim - 1) / blockDim;

	//temp array for pooling result
	result = hipMalloc((float**)&cg->gpuTempLayer, threadCount * sizeof(float));
	assert(result == hipSuccess);


	maxPoolingGPU << <gridDim, blockDim, 0, cg->stream >> > (cg->gpuFeaturePtr, cg->gpuTempLayer, cg->featureWidthSize, cg->featureHeightSize, cg->maskCount, cg->pool, cg->stride);

	cg->featureWidthSize /= cg->stride;
	cg->featureHeightSize /= cg->stride;

	cpuGpuMemCopy(hipMemcpyDeviceToHost, cg, cg->cpuFeaturePtr, cg->gpuTempLayer, threadCount * sizeof(float));

}




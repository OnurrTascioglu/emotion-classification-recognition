#include "hip/hip_runtime.h"
#include <windows.h>
#include <cmath>

#include ""

#include "CpuGpuMem.h"
#include "KernelGpuAdd.cuh"

#define BIAS 1

__global__ void gpu_add(int* gpu_numbers, const int nc)
{
	int id = blockDim.x * blockIdx.x + threadIdx.x;

	if (id < nc)
		gpu_numbers[id] *= 2;
}

__global__ void conv1GPU(float* resultImages, float* masks, BYTE* image, int width, int height, int maskSize, int rMatrixWidth, int rMatrixHeight, int maskCount)
{
	int id = blockDim.x * blockIdx.x + threadIdx.x;
	int temp = 0;
	int k = id % (maskSize * maskSize);
	temp = id / (maskSize * maskSize);
	int j = temp % rMatrixWidth;
	temp = temp / rMatrixWidth;
	int i = temp % rMatrixHeight;
	int m = temp / rMatrixHeight;
	int mCol = k % maskSize;
	int mRow = k / maskSize;

	resultImages[(m * rMatrixWidth * rMatrixHeight) + i * rMatrixWidth + j] +=
		(float)image[(width * i + j) + mRow * width + mCol] * masks[m * (maskSize * maskSize) + k];

	if(id%(maskSize*maskSize) == 0)
		resultImages[(m * rMatrixWidth * rMatrixHeight) + i * rMatrixWidth + j] += (float)BIAS * masks[maskCount * (maskSize * maskSize) + m];

}

__global__ void conv1GPUSetting(float* resultImages, float* masks, BYTE* image, float* weights,int width, int height, int maskSize, int rMatrixWidth, int rMatrixHeight, int maskCount)
{
	int id = blockDim.x * blockIdx.x + threadIdx.x;



	for (int i = 0; i < maskCount * rMatrixWidth * rMatrixHeight; i++) {
		resultImages[i] = 0.0;
	}

	for (int i = 0; i < maskSize * maskSize; i++) {
		for (int j = 0; j < maskCount; j++) {
			masks[j * maskSize * maskSize + i] = weights[i * maskCount + j];
		}
	}

	for (int i = 0; i < maskCount; i++) {
		masks[maskCount * maskSize * maskSize + i] = weights[maskCount * maskSize * maskSize + i];
	}

}

void conv1ExecGPU(CpuGpuMem* cg, BYTE* inputImages,float* weights,const int maskCount, const int imageCount)
{
	int iws = cg->imageWidthSize;
	int ihs = cg->imageHeightSize;
	int ms = cg->maskWHSize;
	int fws = cg->featureWidthSize = iws - ms + 1;
	int fhs = cg->featureHeightSize = ihs - ms + 1;

	int blockDim = 1024;
	int threadCount = maskCount * fws * fhs * ms * ms;




	for (int i = 0; i < maskCount; i++) {
		masks[maskCount * ms * ms + i] = weights[maskCount * ms * ms + i];
	}




	int gridDim = (threadCount + blockDim - 1) / blockDim;

	conv1GPU << <gridDim, blockDim, 0, cg->stream >> > ((int*)cg->gpuPtr, number_count);


	delete[] masks;
	delete[] image;
}





void cpuGpuExecute(CpuGpuMem* cg)
{
	int number_count = cg->allocSize;

	int blockDim = 1024;
	int gridDim = (number_count + blockDim - 1) / blockDim;

	execute
		for (size_t i = 0; i < 4; i++)
			gpu_add << <gridDim, blockDim, 0, cg->stream >> > ((int*)cg->gpuPtr, number_count);
}

//------------------------------------------------------------------
float* conv1(BYTE* inputImage, float* weights, int& width, int& height, int maskSize, int maskCount, int imageCount) {


	int rMatrixWidth = width - maskSize + 1; //extern
	int rMatrixHeight = height - maskSize + 1; //extern


	float* masks = new float[maskSize * maskSize * maskCount + maskCount]; //cpugpuAlloc
	float* resultImages = new float[maskCount * rMatrixWidth * rMatrixHeight]; //cpugpuAlloc
	BYTE* image = new BYTE[width * height]; //cpugpuAlloc



	for (int i = 0; i < width * height; i++) {
		image[i] = inputImage[(imageCount * width * height) + i]; //
	}

	for (int i = 0; i < maskCount * rMatrixWidth * rMatrixHeight; i++) {
		resultImages[i] = 0.0;
	}

	for (int i = 0; i < maskSize * maskSize; i++) {
		for (int j = 0; j < maskCount; j++) {
			masks[j * maskSize * maskSize + i] = weights[i * maskCount + j];
		}
	}
	for (int i = 0; i < maskCount; i++) {
		masks[maskCount * maskSize * maskSize + i] = weights[maskCount * maskSize * maskSize + i]
	}

	for (int m = 0; m < maskCount; m++) {
		for (int i = 0; i < rMatrixHeight; i++) {
			for (int j = 0; j < rMatrixWidth; j++) {
				for (int k = 0; k < maskSize * maskSize; k++) {
					int mCol = k % maskSize;
					int mRow = k / maskSize;
					resultImages[(m * rMatrixWidth * rMatrixHeight) + i * rMatrixWidth + j] +=
						(float)image[(width * i + j) + mRow * width + mCol] * masks[m * (maskSize * maskSize) + k];
				}
				resultImages[(m * rMatrixWidth * rMatrixHeight) + i * rMatrixWidth + j] += (float)BIAS * masks[maskCount * (maskSize * maskSize) + m];
			}
		}
	}

	delete[] masks;
	delete[] image;

	width = width - maskSize + 1;
	height = height - maskSize + 1;

	return resultImages;
}